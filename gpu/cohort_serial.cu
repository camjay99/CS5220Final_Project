#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>
#include <random>
#include <iostream>
#include <fstream>
#include <chrono>
#include <hip/hip_runtime.h>
#include "parameters.h"
#include "radiation_solver.h"
#include "photosynthesis.h"
#include "canopy_air_space.h"

#define NUM_THREADS 256
int blks;

// As a first order operation, we are going to begin with a single cohort and implement a simple
// multi-layer two-stream model that considers one "big flat leaf" and the ground for three different bands. Future versions
// should incorporate additional cohorts that are each treated as seperate "big flat leaves."
// Considerations for evapotranspiration, growth, diurnal cycles, etc. will be implemented in the future.

int num_patches;    // Number of patches to include in a grid cell
int num_cohorts_per_patch;
int seed = 42;
bool print_output = true;

__device__ double dt = 300;

__device__ double* direct_profile_PAR_dev;
__device__ double* direct_profile_NIR_dev;
__device__ double* absorbed_radiance_dev;

double* temp_profile_dev;
double* leaf_area_profile_dev;
double* mass_profile_dev;

// PFT and related stats.
__device__ double wind_speed = 2;
__device__ double air_density = 1.225;
__device__ double G_Wl = 0.01;
__device__ double Vcmax15 = 6.25;
__device__ double Q10Vcmax = 2.40;
__device__ double f_cold = 0.40;
__device__ double f_hot = 0.40;
__device__ double T_cold = 283.15;
__device__ double T_hot = 318.15;
__device__ double f_clump = 0.80;
__device__ double f_R = 0.015;
__device__ double quantum_yield = 0.08;
__device__ double p_c = 1;
__device__ double stomata = 1;
__device__ double soil_temp = 298.15;
__device__ double q_leaf = 3218;
__device__ double leaf_size = 0.1;
__device__ double M = 9;
__device__ double c_c = 400;
__device__ double dw = 0.016;
__device__ double w_c = 0.017;

__device__ double calc_air_temp(double t) {
    return (max_air_temp - min_air_temp) / 2 * std::sin(2 * 3.1415 * t / 86400)  + (max_air_temp + min_air_temp) / 2;
}

__device__ double calc_incoming_PAR(double t) {
    return incoming_direct_PAR / 2 * std::sin(2 * 3.1415 * (t - 7200) / 86400 )  + incoming_direct_PAR / 2;
}

__device__ double calc_incoming_NIR(double t) {
    return incoming_direct_NIR / 2 * std::sin(2 * 3.1415 * (t - 7200) / 86400)  + incoming_direct_NIR/ 2;
}

__device__ double calc_temp_increment(double temp, double incoming_radiation, double incoming_PAR, double air_temp, double leaf_area, double mass) {
    double G_Qlambda = calculate_G_Qlambda(air_temp, temp, leaf_size, wind_speed);
    double G_Wlambda = calculate_G_Wlambda(G_Qlambda, air_density);
    double Q_cohort_canopy = calculate_Q_cohort_canopy(leaf_area, air_density, G_Qlambda, air_temp, temp, w_c);
    double G_Clambda = G_Wlambda/f_Glambda;

    double A = co2_mixing_ratio_solver_C4(G_Wl, G_Wlambda, G_Clambda, M, c_c, w_c, dw, temp, Vcmax15, Q10Vcmax,
                                           f_cold, f_hot, T_cold, T_hot,
                                           incoming_PAR, f_clump, f_R, quantum_yield, p_c);

    double water_flux = calculate_water_flux(stomata, leaf_area, G_Wlambda, G_Wl, w_c, temp, p_c);
    double soil_cohort_enthalpy = calculate_soil_cohort_enthalpy(water_flux, soil_temp);
    double cohort_CAS_enthalpy = calculate_cohort_CAS_enthalpy(water_flux, temp);

    double total_enthalpy_change = (incoming_radiation - Q_cohort_canopy) + (soil_cohort_enthalpy - cohort_CAS_enthalpy);

    return ((total_enthalpy_change) / (mass*((0.7*q_l_water + q_leaf)/1.7)));
}


__global__ void simulate_one_step(int num_patches, int num_cohorts_per_patch, int i, double* temp_profile, double* leaf_area_profile, double* mass_profile) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= num_patches * num_cohorts_per_patch) {
        return;
    }
    //printf("%d ", tid);
    int p = tid / num_cohorts_per_patch;
    int k = tid % num_cohorts_per_patch;
    //printf("%d %d ", p, k);
    //printf("%f ", temp_profile[0]);
        
    //for (int p = 0; p < num_patches; p++) {
        double* direct_profile_PAR = direct_profile_PAR_dev;
        double* direct_profile_NIR = direct_profile_NIR_dev;
        double* absorbed_radiance = absorbed_radiance_dev;
        // Calculate direct radiation profile 
        calculate_direct_profile(direct_profile_PAR, num_cohorts_per_patch, calc_incoming_PAR(i*dt));
        calculate_direct_profile(direct_profile_NIR, num_cohorts_per_patch, calc_incoming_NIR(i*dt));
        calculate_absorbed_radiance(absorbed_radiance, 
                                direct_profile_PAR, direct_profile_NIR,
                                num_cohorts_per_patch);

        // For each cohort in this patch
        //for (int k = 0; k < num_cohorts_per_patch; k++) {
            double k1 = calc_temp_increment(temp_profile[p*num_cohorts_per_patch + k],           absorbed_radiance[k], direct_profile_PAR[k+1] - direct_profile_PAR[k], calc_air_temp(dt*i), 
                                leaf_area_profile[p*num_cohorts_per_patch + k], mass_profile[p*num_cohorts_per_patch + k]);

            double k2 = calc_temp_increment(temp_profile[p*num_cohorts_per_patch + k] + k1*dt/2, absorbed_radiance[k], direct_profile_PAR[k+1] - direct_profile_PAR[k], calc_air_temp(dt*i + dt/2), 
                                leaf_area_profile[p*num_cohorts_per_patch + k], mass_profile[p*num_cohorts_per_patch + k]);

            double k3 = calc_temp_increment(temp_profile[p*num_cohorts_per_patch + k] + k2*dt/2, absorbed_radiance[k], direct_profile_PAR[k+1] - direct_profile_PAR[k], calc_air_temp(dt*i + dt/2), 
                                leaf_area_profile[p*num_cohorts_per_patch + k], mass_profile[p*num_cohorts_per_patch + k]);

            double k4 = calc_temp_increment(temp_profile[p*num_cohorts_per_patch + k] + k3*dt,   absorbed_radiance[k], direct_profile_PAR[k+1] - direct_profile_PAR[k], calc_air_temp(dt*i + dt), 
                                leaf_area_profile[p*num_cohorts_per_patch + k], mass_profile[p*num_cohorts_per_patch + k]);

            temp_profile[p*num_cohorts_per_patch + k] += dt/6 * (k1 + 2*k2 + 2*k3 + k4);

        //}

    //}

}


int main(int argc, char** argv) {
    // As a temporary starting point, we can assume that each patch has ~500 thin cohorts, if we want in the future we can also consider different plant functional types so that parameters can be variable.
    if (argc < 3) {
        printf("Error: please argue num_patches and num_cohorts_per_patch (in that order)\n");
        return -1;
    }
    num_patches = atoi((argv[1]));
    num_cohorts_per_patch = atoi((argv[2]));

    blks = (num_patches * num_cohorts_per_patch + NUM_THREADS - 1) / NUM_THREADS;

    // Initalize patch structure to random values
    std::random_device rd;
    std::mt19937 gen(seed ? seed : rd());
    std::uniform_real_distribution<> la_rd(3., 5.);
    std::uniform_real_distribution<> m_rd(90., 110.);

    double* leaf_area_profile = new double[num_patches*num_cohorts_per_patch];
    double* mass_profile = new double[num_patches*num_cohorts_per_patch];
    for (int k = 0; k < num_patches*num_cohorts_per_patch; k++) {
        leaf_area_profile[k] = la_rd(gen);
        mass_profile[k] = m_rd(gen);
        //printf("%d\t%f\t%f\n", k, leaf_area_profile[k], mass_profile[k]);
    }

    // No need to randomize temp_profiles, as the will quickly converge to long-term behavior.
    double* temp_profile = new double[num_patches*num_cohorts_per_patch];
    
    // Set temperature
    for (int k = 0; k < num_patches*num_cohorts_per_patch; k++) {
        temp_profile[k] = 298.15 + k;
    }


    // alloc GPU space
    hipMalloc((void **)&direct_profile_PAR_dev,       (num_cohorts_per_patch + 1) * sizeof(double) );
    hipMalloc((void **)&direct_profile_NIR_dev,       (num_cohorts_per_patch + 1) * sizeof(double) );
    hipMalloc((void **)&absorbed_radiance_dev ,       (num_cohorts_per_patch)     * sizeof(double) );

    hipMalloc((void **)&temp_profile_dev ,       (num_cohorts_per_patch)     * sizeof(double) );
    hipMemcpy(temp_profile_dev, temp_profile, num_patches * num_cohorts_per_patch * sizeof(double), hipMemcpyHostToDevice);

    hipMalloc((void **)&leaf_area_profile_dev ,       (num_cohorts_per_patch)     * sizeof(double) );
    hipMemcpy(leaf_area_profile_dev, leaf_area_profile, num_patches * num_cohorts_per_patch * sizeof(double), hipMemcpyHostToDevice);

    hipMalloc((void **)&mass_profile_dev ,       (num_cohorts_per_patch)     * sizeof(double) );
    hipMemcpy(mass_profile_dev, mass_profile, num_patches * num_cohorts_per_patch * sizeof(double), hipMemcpyHostToDevice);
	
    // Starting simulation algorithm
    auto start_time = std::chrono::stead_clock::now()

    // for each time step
    for (int i = 0; i < 2016; i++) {

        //printf("%d", i);

        simulate_one_step<<<blks, NUM_THREADS>>>(num_patches, num_cohorts_per_patch, i, temp_profile_dev, leaf_area_profile_dev, mass_profile_dev);

    }

    //printf("\n%d\n", blks);
    //printf("%d\n", NUM_THREADS);

    hipMemcpy(temp_profile, temp_profile_dev, num_patches * num_cohorts_per_patch * sizeof(double), hipMemcpyDeviceToHost); // copy data back from gpu

    printf("\n%f %f %f %f\n", temp_profile[0], temp_profile[1], temp_profile[2], temp_profile[3]);

    auto end_time = std::chrono::steady_clock::now();

    std::chrono::duration<double> diff = end_time - start_time;
    double seconds = diff.count();

    std::cout << "Simulation Time = " << seconds << " seconds for " << num_patches
                  << " patches with " << num_cohorts_per_patch << " cohorts per patch.\n";
    
    return 0;
}
